#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010

#include <stdio.h>

const int N = 16;
const int blocksize = 16;

__global__
void copy(char *a, char *b)
{
        b[threadIdx.x] = a[threadIdx.x];
}


void printDeviceInfo() {
  int nDevices;

  auto err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    printf("hipGetDeviceCount failed, cudaerror: %i\n", err);
    return;
  }
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    auto err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      printf("hipGetDeviceProperties failed, cudaerror: %i\n", err);
      return;
    }
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}


int main()
{
  printDeviceInfo();
        char a[N] = "World\0\0\0\0\0\0\0";
        char b[N] = "Not Working\0\0\0\0";

        char *ad;
        char *bd;
        const int size = N*sizeof(char);

        printf("input: %s\n", a);
        hipError_t err;
        err = hipMalloc( (void**)&ad, size );
	if (err != hipSuccess) {
	  printf("cudaerror: %i\n", err);
//	  return EXIT_FAILURE;
	}

        hipMalloc( (void**)&bd, size );
        hipMemcpy( ad, a, size, hipMemcpyHostToDevice );
        hipMemcpy( bd, b, size, hipMemcpyHostToDevice );

        dim3 dimBlock( blocksize, 1 );
        dim3 dimGrid( 1, 1 );
        copy<<<dimGrid, dimBlock>>>(ad, bd);
        hipMemcpy( b, bd, size, hipMemcpyDeviceToHost );
        hipFree( ad );
        hipFree( bd );

        printf("Hello %s\n", b);
	if (strncmp(a, b, N)) {
	  printf("cuda kernel did not return expected result\n");
	  return EXIT_FAILURE;
	}
        return EXIT_SUCCESS;
}

